#include <stdio.h> 

int main() {
  int nDevices;
 


//All CUDA C Runtime API functions have a return value which can be used to check for errors that occurr during their execution

  //hipPeekAtLastError(): cuda maintain a single variable for error, which is updated everytime. This method will return the value of this variable
  //hipGetLastError(): this does the same as above function do, but after fetching the value it resets it to hipSuccess
  //hipDeviceSynchronize(): this check the device async errors cause by the issuing command to the device from host, this can be achieved by doing folowing as well
  //    if (errAsync != hipSuccess)
  //        printf("Async kernel error: %s\n", hipGetErrorString(hipGetLastError());

hipError_t err = hipGetDeviceCount(&nDevices);
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));

//  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}
